/**
 * Copyright 2020 Sajeeb Roy Chowdhury
 *
 * Permission is hereby granted, free of charge, to any person
 * obtaining a copy of this software and associated documentation
 * files (the "Software"), to deal in the Software without
 * restriction, including without limitation the rights to use,
 * copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software
 * is furnished to do so, subject to the following conditions:
 * The above copyright notice and this permission notice shall be
 * included in all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
 * IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY
 * CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,
 * TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
 * SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */

#include <hip/hip_runtime.h>

__device__ __host__
int remainder_is_nonzero(const int& da, bool* A, const int& db, const uint64_t& B)
// returns true if the remainder of A after division by B is nonzero
{
	for (int i = da + db; i >= db; i--) {
		if (A[i]) {
			for (int j = db, k = i; j >= 0; j--, k--) {
				A[k] = (A[k] + ((B >> (db-j))&1)) & 1;
			}
		}
	}
	for (int k = da + db; k >= 0; k--) {
		if (A[k]) {
			return true;
		}
	}
	return false;
}

template<int da, int dc>
__device__ __host__
bool test_all_two_bit_patterns(const uint64_t& C)
// returns true if division by C leaves a nonzero remainder for all two bit error patters
{
	bool B[da + dc + 1];
	bool A[da + 1 + dc];

	memset(A, 0, da + dc + 1);
	memset(B, 0, da + dc + 1);

	for (int i = 0; i <= da; i++) {
		for (int ai = 1; ai < 2; ai++) {
			A[i] = ai;
			for (int j = i + 1; j <= da; j++) {
				for (int aj = 1; aj < 2; aj++) {
					A[j] = aj;
					for (int k = 0; k <= da; k++) {
						B[dc + k] = A[k];
					}
					for (int k = 0; k < dc; k++) {
						B[k] = 0;
					}
					if (!remainder_is_nonzero (da, B, dc, C)) {
						return false;
					}
				}
				A[j] = 0;
			}
		}
		A[i] = 0;
	}
	return true;
}

template<int da, int dc>
__device__ __host__
bool test_all_three_bit_patterns(const uint64_t& C)
// returns true if division by C leaves a nonzero remainder for all two bit error patters
{
	bool B[da + dc + 1];
	bool A[da + 1 + dc];

	memset(A, 0, da + dc + 1);
	memset(B, 0, da + dc + 1);

	for (int i1 = 0; i1 <= da; i1++) {
		for (int a1 = 1; a1 < 2; a1++) {
			A[i1] = a1;
			for (int i2 = i1 + 1; i2 <= da; i2++) {
				for (int a2 = 1; a2 < 2; a2++) {
					A[i2] = a2;
					for (int i3 = i2 + 1; i3 <= da; i3++) {
						for (int a3 = 1; a3 < 2; a3++) {
							A[i3] = a3;
							for (int h = 0; h <= da; h++) {
								B[dc + h] = A[h];
							}
							for (int h = 0; h < dc; h++) {
								B[h] = 0;
							}
							if (!remainder_is_nonzero (da, B, dc, C)) {
								return false;
							}
						}
						A[i3] = 0;
					}
				}
				A[i2] = 0;
			}
		}
		A[i1] = 0;
	}
	return true;
}

template<int da, int dc>
__global__
void CRC_polynomial_cuda_t2(uint64_t C, uint64_t e, bool* res) {
	uint64_t thread_id = threadIdx.x + blockIdx.x * blockDim.x;
	bool ret = false;
	res[thread_id] = ret;

	C += thread_id;

	if (!(C&1)) return;
	if (C >= e) return;
	if (C > (uint64_t(1)<<(dc+1))-1) return;

	ret = test_all_two_bit_patterns<da, dc>(C);
	res[thread_id] = ret;
}

template<int da, int dc>
__global__
void CRC_polynomial_cuda_t3(uint64_t* data, bool* res, size_t size) {
	uint64_t thread_id = threadIdx.x + blockIdx.x * blockDim.x;
	if (thread_id > size) return;
	res[thread_id] = false;
	res[thread_id] = test_all_three_bit_patterns<da, dc>(data[thread_id]);
}
