#include "hip/hip_runtime.h"
/**
 * Copyright 2020 Sajeeb Roy Chowdhury
 *
 * Permission is hereby granted, free of charge, to any person
 * obtaining a copy of this software and associated documentation
 * files (the "Software"), to deal in the Software without
 * restriction, including without limitation the rights to use,
 * copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software
 * is furnished to do so, subject to the following conditions:
 * The above copyright notice and this permission notice shall be
 * included in all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
 * IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY
 * CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,
 * TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
 * SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */

#include <hip/hip_runtime.h>

__device__ __host__
int remainder_is_nonzero(const int& da, uint32_t* A, const int& db, const uint64_t& B)
// returns true if the remainder of A after division by B is nonzero
{
	const int size = da + db + 1;
	for (int i = da + db; i >= db; i--) {
		if (A[i/size]&(uint32_t(1)<<(i & 31))) {
			for (int j = db, k = i; j >= 0; j--, k--) {
				int& ak = A[k/size];
				int& b = (B >> (db-j))&1;
				int& v = (((ak>>(k&31))&1)^b);
				ak &= ~(uint32_t(1) << (k&31));
				ak |= v << (k&31);
			}
		}
	}
	for (int k = da + db; k >= 0; k--) {
		if (A[k/size]&(uint32_t(1) << (k&31))) {
			return true;
		}
	}
	return false;
}

template<int da, int dc>
__device__ __host__
bool test_all_two_bit_patterns(const uint64_t& C)
// returns true if division by C leaves a nonzero remainder for all two bit error patters
{
	constexpr int size = da + dc + 1;
	constexpr int arr_size = 1+ size/32;
	uint32_t B[arr_size];
	uint32_t A[arr_size];
	memset(A, 0, da + dc + 1);
	memset(B, 0, da + dc + 1);

	for (int i = 0; i <= da; i++) {
		for (int ai = 1; ai < 2; ai++) {
			A[i/32] |= uint32_t(1) << (i & 31);
			for (int j = i + 1; j <= da; j++) {
				for (int aj = 1; aj < 2; aj++) {
					A[j/32] |= uint32_t(1) << (j & 31);
					for (int k = 0; k <= da; k++) {
						int& b = B[(dc + k)/32];
						int& a = A[k/32];
//						B[dc + k] = A[k];
						b &= ~(uint32_t(1) << ((dc+k)&31));
						b |= (a & (uint32_t(1) << (k & 31))) << (dc & 31);
					}
					for (int k = 0; k < dc; k++) {
//						B[k] = 0;
						B[k/32] &= ~(uint32_t(1) << (k & 31));
					}
					if (!remainder_is_nonzero (da, B, dc, C)) {
						return false;
					}
				}
				A[j/32] &= ~(uint32_t(1) << (j & 31));
			}
		}
		A[i/32] &= ~(uint32_t(1) << (i & 31));
	}
	return true;
}

template<int da, int dc>
__device__ __host__
bool test_all_three_bit_patterns(const uint64_t& C)
// returns true if division by C leaves a nonzero remainder for all two bit error patters
{
	constexpr int size = da + dc + 1;
	constexpr int arr_size = 1+ size/32;
	uint32_t B[arr_size];
	uint32_t A[arr_size];
	memset(A, 0, da + dc + 1);
	memset(B, 0, da + dc + 1);

	for (int i1 = 0; i1 <= da; i1++) {
		for (int a1 = 1; a1 < 2; a1++) {
			A[i1/32] |= uint32_t(1) << (i1 & 31);
			for (int i2 = i1 + 1; i2 <= da; i2++) {
				for (int a2 = 1; a2 < 2; a2++) {
					A[i2/32] |= uint32_t(1) << (i2 & 31);
					for (int i3 = i2 + 1; i3 <= da; i3++) {
						for (int a3 = 1; a3 < 2; a3++) {
							A[i3/32] |= uint32_t(1) << (i3 & 31);
							for (int h = 0; h <= da; h++) {
								int& b = B[(dc+h)/32];
								int& a = A[h/32];
								b &= ~(uint32_t(1) << ((dc+h) & 31));
								b |= (a & (uint32_t(1) << (h & 31))) << (dc & 31);
							}
							for (int h = 0; h < dc; h++) {
//								B[h] = 0;
								B[h/32] &= ~(uint32_t(1) << (h & 31));
							}
							if (!remainder_is_nonzero (da, B, dc, C)) {
								return false;
							}
						}
						A[i3/32] &= ~(uint32_t(1) << (i3&31));
					}
				}
				A[i2/32] &= ~(uint32_t(1) << (i2&31));
			}
		}
		A[i1/32] &= ~(uint32_t(1) << (i1&31));
	}
	return true;
}

template<int da, int dc>
__global__
void CRC_polynomial_cuda_t2(uint64_t C, uint64_t e, bool* res) {
	uint64_t thread_id = threadIdx.x + blockIdx.x * blockDim.x;
	bool ret = false;
	res[thread_id] = ret;

	C += thread_id;

	if (!(C&1)) return;
	if (C >= e) return;
	if (C > (uint64_t(1)<<(dc+1))-1) return;

	ret = test_all_two_bit_patterns<da, dc>(C);
	res[thread_id] = ret;
}

template<int da, int dc>
__global__
void CRC_polynomial_cuda_t3(uint64_t* data, bool* res, size_t size) {
	uint64_t thread_id = threadIdx.x + blockIdx.x * blockDim.x;
	if (thread_id > size) return;
	res[thread_id] = test_all_three_bit_patterns<da, dc>(data[thread_id]);
}
