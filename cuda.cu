/**
 * Copyright 2020 Sajeeb Roy Chowdhury
 *
 * Permission is hereby granted, free of charge, to any person
 * obtaining a copy of this software and associated documentation
 * files (the "Software"), to deal in the Software without
 * restriction, including without limitation the rights to use,
 * copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software
 * is furnished to do so, subject to the following conditions:
 * The above copyright notice and this permission notice shall be
 * included in all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
 * IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY
 * CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,
 * TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
 * SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */

#include <hip/hip_runtime.h>

__device__ __host__
int remainder_is_nonzero(const int& da, uint32_t* A, const int& db, const uint64_t& B)
// returns true if the remainder of A after division by B is nonzero
{
	for (int i = da + db; i >= db; i--) {
		if (A[i/32] & (1 << (i&31))) {
			for (int j = db, k = i; j >= 0; j--, k--) {
				const uint32_t& b = uint32_t((B >> (db-j)) & 1);
				const uint32_t& a = (A[k/32] >> (k&31)) & 1;
				A[k/32] &= ~(1 << (k & 31));
				A[k/32] |= (a ^ b) << (k & 31);
			}
		}
	}
	for (int k = da + db; k >= 0; k--) {
		if (A[k/32] & (1 << (k & 31))) {
			return true;
		}
	}
	return false;
}

template<int da, int dc>
__device__ __host__
bool test_all_two_bit_patterns(const uint64_t& C)
// returns true if division by C leaves a nonzero remainder for all two bit error patters
{
	constexpr int size = da + dc + 1;
	uint32_t B[size/32+1];
	uint32_t A[size/32+1];

	memset(A, 0, 4 * (size/32+1));
	memset(B, 0, 4 * (size/32+1));

	for (int i = 0; i <= da; i++) {
		A[i/32] |= 1u << (i&31);
		for (int j = i + 1; j <= da; j++) {
			A[j/32] |= 1u << (j&31);
			for (int k = 0; k <= da; k++) {
//				B[dc + k] = A[k];
				B[(dc+k)/32] &= ~(1u << ((dc+k) & 31));
				B[(dc+k)/32] |= ((A[k/32] >> (k&31)) & 1) << ((dc+k) & 31);
			}
			for (int k = 0; k < dc; k++) {
//				B[k] = 0;
				B[k/32] &= ~(1 << (k&31));
			}
			if (!remainder_is_nonzero (da, B, dc, C)) {
				return false;
			}
			A[j/32] &= ~(1 << (j&31));
		}
		A[i/32] &= ~(1 << (i&31));
	}
	return true;
}

template<int da, int dc>
__device__ __host__
bool test_all_three_bit_patterns(const uint64_t& C)
// returns true if division by C leaves a nonzero remainder for all two bit error patters
{
	constexpr int size = da + dc + 1;
	uint32_t B[size/32+1];
	uint32_t A[size/32+1];

	memset(A, 0, 4 * (size/32+1));
	memset(B, 0, 4 * (size/32+1));

	for (int i1 = 0; i1 <= da; i1++) {
		for (int a1 = 1; a1 < 2; a1++) {
			A[i1/32] |= 1u << (i1 & 31);
			for (int i2 = i1 + 1; i2 <= da; i2++) {
				for (int a2 = 1; a2 < 2; a2++) {
					A[i2/32] |= 1u << (i2 & 31);
					for (int i3 = i2 + 1; i3 <= da; i3++) {
						for (int a3 = 1; a3 < 2; a3++) {
							A[i3/32] |= 1u << (i3 & 31);
							for (int h = 0; h <= da; h++) {
								// B[dc + h] = A[h];
								B[(dc+h)/32] &= ~(1u << ((dc+h) & 31));
								B[(dc+h)/32] |= ((A[h/32] >> (h & 31)) & 1) << ((dc+h) & 31);
							}
							for (int h = 0; h < dc; h++) {
								// B[h] = 0;
								B[h/32] &= ~(1 << (h & 31));
							}
							if (!remainder_is_nonzero (da, B, dc, C)) {
								return false;
							}
						}
						A[i3/32] &= ~(1u << (i3 & 31));
					}
				}
				A[i2/32] &= ~(1u << (i2 & 31));
			}
		}
		A[i1/32] &= ~(1u << (i1 & 31));
	}
	return true;
}

template<int da, int dc>
__global__
void CRC_polynomial_cuda_t2(uint64_t C, uint64_t e, bool* res) {
	uint64_t thread_id = threadIdx.x + blockIdx.x * blockDim.x;
	bool ret = false;
	res[thread_id] = ret;

	C += thread_id;

	if (!(C&1)) return;
	if (C >= e) return;
	if (C > (uint64_t(1)<<(dc+1))-1) return;

	ret = test_all_two_bit_patterns<da, dc>(C);
	res[thread_id] = ret;
}

template<int da, int dc>
__global__
void CRC_polynomial_cuda_t3(uint64_t* data, bool* res, size_t size) {
	uint64_t thread_id = threadIdx.x + blockIdx.x * blockDim.x;
	if (thread_id > size) return;
	res[thread_id] = false;
	res[thread_id] = test_all_three_bit_patterns<da, dc>(data[thread_id]);
}
