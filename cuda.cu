/**
 * Copyright 2020 Sajeeb Roy Chowdhury
 *
 * Permission is hereby granted, free of charge, to any person
 * obtaining a copy of this software and associated documentation
 * files (the "Software"), to deal in the Software without
 * restriction, including without limitation the rights to use,
 * copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software
 * is furnished to do so, subject to the following conditions:
 * The above copyright notice and this permission notice shall be
 * included in all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
 * IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY
 * CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,
 * TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
 * SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */

 #include <hip/hip_runtime.h>

 __device__ __host__
 int remainder_is_nonzero(const int& da, bool* A, const int& db, const uint64_t& B)
 // returns true if the remainder of A after division by B is nonzero
 {
	 for (int i = da + db; i >= db; i--) {
		const bool& ai = A[i];
		if (ai) 
			for (int j = db, k = i; j >= 0; j--, k--) {
				A[k] = (A[k]^((B >> (db-j))&1));
			}
	 }
	 for (int k = da + db; k >= 0; k--) {
		 if (A[k]) {
			 return true;
		 }
	 }
	 return false;
 }
 
 template<int da, int dc>
 __device__ __host__
 bool test_all_two_bit_patterns(const uint64_t& C)
 // returns true if division by C leaves a nonzero remainder for all two bit error patters
 {
	 bool B[da + dc + 1];
	 bool A[da + 1 + dc];
 
	 memset(A, 0, da + dc + 1);
	 memset(B, 0, da + dc + 1);
 
	 for (int i = 0; i <= da; i++) {
		A[i] = 1;
		for (int j = i + 1; j <= da; j++) {
			A[j] = 1;
			for (int k = 0; k <= da; k++) B[dc + k] = A[k];
			for (int k = 0; k < dc; k++) B[k] = 0;
			if (!remainder_is_nonzero (da, B, dc, C)) return false;
			#if __CUDA_ARCH__
			__syncthreads();
			#endif
			A[j] = 0;
		}
		A[i] = 0;
	 }
	 return true;
 }
 
 template<int da, int dc>
 __device__ __host__
 bool test_all_three_bit_patterns(const uint64_t& C)
 // returns true if division by C leaves a nonzero remainder for all two bit error patters
 {
	 bool B[da + dc + 1];
	 bool A[da + 1 + dc];
 
	 memset(A, 0, da + dc + 1);
	 memset(B, 0, da + dc + 1);
 
	 for (int i1 = 0; i1 <= da; i1++) {
		A[i1] = 1;
		for (int i2 = i1 + 1; i2 <= da; i2++) {
			A[i2] = 1;
			for (int i3 = i2 + 1; i3 <= da; i3++) {
				A[i3] = 1;
				for (int h = 0; h <= da; h++) B[dc + h] = A[h];
				for (int h = 0; h < dc; h++) B[h] = 0;
				if (!remainder_is_nonzero (da, B, dc, C)) return false;
				#if __CUDA_ARCH__
				__syncthreads();
				#endif
				A[i3] = 0;
			}
			A[i2] = 0;
		}
		A[i1] = 0;
	 }
	 return true;
 }
 
 template<int da, int dc>
 __global__
 void CRC_polynomial_cuda_t2(uint64_t C, uint64_t e, bool* res) {
	 uint64_t thread_id = threadIdx.x + blockIdx.x * blockDim.x;
	 bool ret = false;
	 res[thread_id] = ret;
 
	 C += thread_id;
 
	 if (!(C&1)) return;
	 if (C >= e) return;
	 if (C > (uint64_t(1)<<(dc+1))-1) return;
 
	 ret = test_all_two_bit_patterns<da, dc>(C);
	 res[thread_id] = ret;
 }
 
 template<int da, int dc>
 __global__
 void CRC_polynomial_cuda_t3(uint64_t* data, bool* res, size_t size) {
	 uint64_t thread_id = threadIdx.x + blockIdx.x * blockDim.x;
	 if (thread_id > size) return;
	 res[thread_id] = test_all_three_bit_patterns<da, dc>(data[thread_id]);
 }